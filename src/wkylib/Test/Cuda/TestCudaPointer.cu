#include "hip/hip_runtime.h"
#include <wkylib/Test/catch.hpp>
#include <wkylib/Cuda/CudaPointer.h>

using namespace WKYLIB::Cuda;

TEST_CASE( "CudaPointer initialization", "[CudaPointer]" ) {
    int test = 20;

    SECTION("initialize with value")
    {
        CudaPointer<int> p(test);
        REQUIRE(*p == test);
    }

    SECTION("initialize with assign()")
    {
        CudaPointer<int> p;
        p.assign(test);
        REQUIRE(*p == test);
    }
}

class TestObject
{
public:
    CudaPointer<int> data;
};

__global__ void kernel_test_cuda_manipulation(CudaPointer<int> p)
{
    *p = 2;
}

__global__ void kernel_test_cuda_manipulation_with_object(CudaPointer<TestObject> p)
{
    *p->data = 2;
}

TEST_CASE( "CudaPointer manipulation with gpu kernel", "[CudaPointer]" ) {
    int test = 20;
    TestObject testObject;
    testObject.data.assign(test);

    CudaPointer<int> p(test);
    kernel_test_cuda_manipulation <<<1, 1>>> (p);
    hipDeviceSynchronize();
    REQUIRE(*p == 2);

    CudaPointer<TestObject> p2(testObject);
    kernel_test_cuda_manipulation_with_object <<<1, 1>>> (p2);
    hipDeviceSynchronize();
    REQUIRE(*p2->data == 2);
}

__global__ void kernel_test_nestification(CudaPointer<CudaPointer<int>> p)
{
    **p = 2;
}

TEST_CASE( "CudaPointer multiple nestification", "[CudaPointer]" ) {
    CudaPointer<int> gpu_int;
    gpu_int.assign(1);
    CudaPointer<CudaPointer<int>> gpu_gpu_int;
    gpu_gpu_int.assign(gpu_int);
    REQUIRE(**gpu_gpu_int == 1);

    kernel_test_nestification <<<1, 1>>> (gpu_gpu_int);
    hipDeviceSynchronize();

    REQUIRE(**gpu_gpu_int == 2);
}

